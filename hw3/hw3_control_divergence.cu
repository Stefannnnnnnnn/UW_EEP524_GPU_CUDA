#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void noDivergence(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void singleBranch(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    if (threadIdx.x % 32 < 16) {
        c[i] = a[i] + b[i];
    }
    else {
        c[i] = a[i] * b[i];
    }
}

__global__ void nestedBranch(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    if (threadIdx.x % 32 < 8) {
        c[i] = a[i] + b[i];
    }
    else if (threadIdx.x % 32 < 8){
        c[i] = a[i] + 2 * b[i];
    }
    else if (threadIdx.x % 32 < 16) {
        c[i] = a[i] + 3 * b[i];
    }
    else if (threadIdx.x % 32 < 24) {
        c[i] = a[i] + 4 * b[i];
    }
}

int main()
{
    const int arraySize = 32 * 1e5;
    int* a = (int*)malloc(arraySize * sizeof(int));
    int* b = (int*)malloc(arraySize * sizeof(int));
    int* c = (int*)malloc(arraySize * sizeof(int));

    for (int x = 0, int* pa = a, int* pb = b; x < arraySize; x++)
    {
        *pa++ = x + 1;
        *pb++ = 2 * (x + 1);
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    dim3 threadPerBlock(32, 1, 1);
    dim3 blockPerGrid((size + threadPerBlock.x - 1) / threadPerBlock.x, 1, 1);
    // Launch a kernel on the GPU with one thread for each element.
    noDivergence << <blockPerGrid, threadPerBlock >> > (dev_c, dev_a, dev_b);
    cudaStatus = hipDeviceSynchronize();
    singleBranch << <blockPerGrid, threadPerBlock >> > (dev_c, dev_a, dev_b);
    cudaStatus = hipDeviceSynchronize();
    nestedBranch << <blockPerGrid, threadPerBlock >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
